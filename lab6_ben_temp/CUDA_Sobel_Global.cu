#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdint.h>
#include<time.h>
#include<stdio.h>
#include<hip/hip_runtime_api.h>

#define BLOCK_DIM_X 100
#define BLOCK_DIM_Y 10
#define GRID_DIM_X 500
#define GRID_DIM_Y 50
#define N 5000

// Forward declaration of the kernel function
__global__ void SobelFilter(uint8_t*, uint8_t*);

// Kernel (Runs on Device)
__global__ void SobelFilter(uint8_t* A, uint8_t* B) //You may add more variables to help with writing your code
{
 
// REQD: Find Global Index (Hint: You may use the thread index cheat sheet)
// Start
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

// End

 
// REQD: Write the Kernel Code
// Note: Remember to define all the filter values here (So that it is saved on to the register)
// Start
const int8_t sobel_x[3][3] = {{-1, 0, 1},
                               {-2, 0, 2},
                               {-1, 0, 1}};
const int8_t sobel_y[3][3] = {{-1, -2, -1},
                               { 0,  0,  0},
                               { 1,  2,  1}};

int16_t gradient_x = 0;
int16_t gradient_y = 0;

// if the pixel is on the boundary of the image, skip the calculation...
if (
    index < N || \ // top row
    index >= N * N - N || \ // bottom row
    index % N == 0 || \ // left column
    index % N == N -1 // right column
) {
    B[index] = 0;
} else {
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            gradient_x += A[index + (i * N) + j] * sobel_x[i + 1][j + 1];
            gradient_y += A[index + (i * N) + j] * sobel_y[i + 1][j + 1];
            }
        }
        int16_t gradient = abs(gradient_x) + abs(gradient_y);
    B[index] = (gradient > 255) ? 255 : gradient;
}

// End

}

// Function to read image
__host__ void read_image_file(const char *filename, uint8_t* image) {

    FILE *infile;

    if ((infile = fopen(filename, "r")) == NULL) {
        fprintf(stderr, "Can't open %s\n", filename);
        exit(1);
    }

    for (int y = 0; y < N; y++) {
        for (int x = 0; x < N; x++) {
            fscanf(infile, "%d", &image[y * N + x]);
            // Read and discard the tab or newline character
            char c;
            fscanf(infile, "%c", &c);
	    }
    }

    fclose(infile);
}

// Function to write image
__host__ void write_image_file(const char *filename, uint8_t* image) {
    FILE *outfile;

    if ((outfile = fopen(filename, "w")) == NULL) {
        fprintf(stderr, "Can't open %s\n", filename);
        exit(1);
    }

    for (int y = 0; y < N; y++) {
        for (int x = 0; x < N; x++) {
	        fprintf(outfile, "%d", image[y * N + x]);
            if (x < N - 1) {
                fprintf(outfile, "\t"); // Delimit with tabs within a row
            }
	    }
        fprintf(outfile, "\n"); // Newline between rows
    }

    fclose(outfile);
}

// Main Function (Runs on Host)
int main(int argc, char ** argv)
{

// Step 1: Allocate Host Memory (You may use malloc() to fix any segmentation errors in this part)
int size = N*N*sizeof(uint8_t);
uint8_t* A = (uint8_t*)malloc(size);
uint8_t* B = (uint8_t*)malloc(size);

if (A == NULL || B == NULL) {
    printf("Memory allocation failed.\n");
    return 1;
}

clock_t before_init = clock();
// REQD: Initialize the value of input matrix A (load the values)
// Start
read_image_file(argv[1], A);

// End


for (int i=0; i < N; i++){
	for (int j=0; j < N; j++){
		B[i*N+j] = 0;
	}
}


clock_t after_init = clock();

hipProfilerStart();
// REQD: Step 2: Allocate device memory for A and B
// Start
uint8_t* d_A;
hipError_t err = hipMalloc((void**)&d_A, size);
printf("CUDA malloc d_A: %s\n",hipGetErrorString(err));
uint8_t* d_B;
err = hipMalloc((void**)&d_B, size);
printf("CUDA malloc d_B: %s\n",hipGetErrorString(err));

// End

// Step 3: Copy data from host memory to device memory
hipEvent_t start_memcpyh2d,stop_memcpyh2d;
hipEventCreate(&start_memcpyh2d);
hipEventCreate(&stop_memcpyh2d);


hipEventRecord(start_memcpyh2d);
// REQD: Code to Copy Data from Host to Device
// Start
err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
printf("CUDA Memcpy A->Ad: %s\n",hipGetErrorString(err));

printf("MEMCPY of %d bytes from Host to Device \n",size);

// End
hipEventRecord(stop_memcpyh2d);

float mseconds1 = 0.0;
hipEventElapsedTime(&mseconds1,start_memcpyh2d,stop_memcpyh2d);
printf("Time of the MEMCPY of %d bytes: %2.3f ms\n",size,mseconds1);


// REQD: Setup the execution configuration (Use dim3 objects to setup the Blocks and Grid)
// Start

dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y);
dim3 dimGrid(GRID_DIM_X, GRID_DIM_Y);


// End

// Step 4: Launch the device computation
hipEvent_t start_kernel,stop_kernel;
hipEventCreate(&start_kernel);
hipEventCreate(&stop_kernel);

hipEventRecord(start_kernel);
// REQD: Write the kernel call with <<< >>> to launch the GPU device
// Start
SobelFilter<<<dimGrid, dimBlock>>>(d_A, d_B);

// End
hipEventRecord(stop_kernel);

// Step 4.5: Wait for the Kernel to complete the workload
hipError_t errk = hipDeviceSynchronize();
hipEventSynchronize(stop_kernel);

float mksec = 0.0;
hipEventElapsedTime(&mksec,start_kernel,stop_kernel);
printf("Time to complete CUDA Sobel kernel %d size: %2.3f ms\n",N,mksec);
printf("CUDA kernel launch: %s\n",hipGetErrorString(errk));


// Step 5: Read results (matrix B) from the device
hipEvent_t start_memcpyd2h,stop_memcpyd2h;
hipEventCreate(&start_memcpyd2h);
hipEventCreate(&stop_memcpyd2h);
hipEventRecord(start_memcpyd2h);
// REQD: Write the code to copy result back to the Host (CPU)
// Start
err = hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
printf("CUDA Memcpy d_B->B: %s\n",hipGetErrorString(err));

// End
hipEventRecord(stop_memcpyd2h);

float mseconds2 = 0.0;
hipEventElapsedTime(&mseconds2,start_memcpyd2h,stop_memcpyd2h);
printf("Time of the MEMCPY of %d bytes : %2.3f ms\n",size,mseconds2);

// REQD: Step 6: Free device memory
// Start
hipFree(d_A);
hipFree(d_B);

// End

hipProfilerStop();

printf("Execution time for initialization(msec) = %d\n",(((after_init-before_init)*1000)/CLOCKS_PER_SEC));
printf("Execution time for CUDA Sobel Filter(msec)= %2.3f\n",mksec);

write_image_file(argv[2], B);

// Free host memory
free(A);
free(B);

return 0;
}	







