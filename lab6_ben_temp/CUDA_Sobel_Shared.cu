
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdint.h>
#include<time.h>
#include<stdio.h>
#include<hip/hip_runtime_api.h>

#define BLOCK_DIM_X 10
#define BLOCK_DIM_Y 10
#define GRID_DIM_X 500
#define GRID_DIM_Y 500
#define N 5000

// Forward declaration of the kernel function
__global__ void SobelFilter(uint8_t*, uint8_t*);

// Kernel (Runs on Device)
__global__ void SobelFilter(uint8_t* A, uint8_t* B) //You may add more variables to help with writing your code
{
 
// REQD: Find Global Index (Hint: You may use the thread index cheat sheet)
// Start
int blockId = blockIdx.x + blockIdx.y * gridDim.x;

int width = gridDim.x * blockDim.x;

int bx = blockDim.x;
int by = blockDim.y;

int tx = threadIdx.x;
int ty = threadIdx.y;

int column = blockIdx.x * bx + tx;
int row = blockIdx.y * by + ty; 

int isLeftEdge = 0;
int isRightEdge = 0;
int isTopEdge = 0;
int isBottomEdge = 0;

// End

// REQD: Define shared memory (**NEW**)
// Start

__shared__ uint8_t local_A[BLOCK_DIM_Y+2][BLOCK_DIM_X+2];

// every thread copies its center pixel
local_A[ty+1][tx+1] = A[row * width + column];
// some threads copy edges/corners
{

    isTopEdge = (ty == 0) && (row != 0);
    isBottomEdge = (ty == BLOCK_DIM_Y-1) && (row != N - 1);

	isLeftEdge = (tx == 0) && (column != 0);
    isRightEdge = (tx == BLOCK_DIM_X-1) && (column != N - 1);

	// init 4 corners
	if (isTopEdge && isLeftEdge) {
		local_A[0][0] = A[(row - 1) * width + (column - 1)];
    } else if (isTopEdge && isRightEdge) {
        local_A[0][BLOCK_DIM_X+1] = A[(row -1) * width + column + 1];
    } else if (isBottomEdge && isLeftEdge) {
	    local_A[BLOCK_DIM_Y+1][0] = A[(row + 1) * width + column - 1];
    } else if (isBottomEdge && isRightEdge) {
	    local_A[BLOCK_DIM_Y+1][BLOCK_DIM_X+1] = A[(row + 1) * width + column + 1];
	}

	// init 4 edges
    if (isTopEdge)  {
        local_A[0][tx+1] = A[(row - 1) * width + column];
    }
    if (isLeftEdge) {
        local_A[ty+1][0] = A[(row * width) + column - 1];
    }
    if (isBottomEdge) {
        local_A[BLOCK_DIM_Y+1][tx+1] = A[(row * width) + column];
    }
    if (isRightEdge) {
	    local_A[ty+1][BLOCK_DIM_X+1] = A[(row * width) + column + 1];
    }

}

__syncthreads();

// DEBUG
// if (blockId == 100000 && tx == 0 && ty == 0) {
//     for (int i = 0; i < BLOCK_DIM_X + 2; i++) {
//         for (int j = 0; j < BLOCK_DIM_Y + 2; j++) {
//             printf("%d\t", local_A[i][j]);
//         }
//         printf("\n");
//     }
// }

if (row == 0 || column == 0 || row == N - 1 || column == N - 1) {
    B[row * width + column] = 0;
    return;
}

// End

// REQD: Write the Kernel Code
// Note: Remember to define all the filter values here (So that it is saved on to the register)
// Start
const int8_t sobel_x[3][3] = {{-1, 0, 1},
                               {-2, 0, 2},
                               {-1, 0, 1}};
const int8_t sobel_y[3][3] = {{-1, -2, -1},
                               { 0,  0,  0},
                               { 1,  2,  1}};

int16_t gradient_x = 0;
int16_t gradient_y = 0;

for (int i = -1; i <= 1; i++) {
    for (int j = -1; j <= 1; j++) {
        gradient_x += local_A[ty+1+i][tx+1+j] * sobel_x[i + 1][j + 1];
        gradient_y += local_A[ty+1+i][tx+1+j] * sobel_y[i + 1][j + 1];
    }
}

int16_t gradient = abs(gradient_x) + abs(gradient_y);
B[row * width + column] = (gradient > 255) ? 255 : gradient;

// End

}

// Function to read image
__host__ void read_image_file(const char *filename, uint8_t* image) {

    FILE *infile;

    if ((infile = fopen(filename, "r")) == NULL) {
        fprintf(stderr, "Can't open %s\n", filename);
        exit(1);
    }

    for (int y = 0; y < N; y++) {
        for (int x = 0; x < N; x++) {
            fscanf(infile, "%d", &image[y * N + x]);
            // Read and discard the tab or newline character
            char c;
            fscanf(infile, "%c", &c);
	    }
    }

    fclose(infile);
}

// Function to write image
__host__ void write_image_file(const char *filename, uint8_t* image) {
    FILE *outfile;

    if ((outfile = fopen(filename, "w")) == NULL) {
        fprintf(stderr, "Can't open %s\n", filename);
        exit(1);
    }

    for (int y = 0; y < N; y++) {
        for (int x = 0; x < N; x++) {
	        fprintf(outfile, "%d", image[y * N + x]);
            if (x < N - 1) {
                fprintf(outfile, "\t"); // Delimit with tabs within a row
            }
	    }
        fprintf(outfile, "\n"); // Newline between rows
    }

    fclose(outfile);
}

// Main Function (Runs on Host)
int main(int argc, char ** argv)
{

if (argc != 3) {
    printf("Usage: %s <input_image.jpg> <output_image.jpg>\n", argv[0]);
    return 1;
}

// Step 1: Allocate Host Memory (You may use malloc() to fix any segmentation errors in this part)
int size = N*N*sizeof(uint8_t);
uint8_t* A = (uint8_t*)malloc(size);
uint8_t* B = (uint8_t*)malloc(size);

if (A == NULL || B == NULL) {
    printf("Memory allocation failed.\n");
    return 1;
}

clock_t before_init = clock();
// REQD: Initialize the value of input matrix A (load the values)
// Start
read_image_file(argv[1], A);

// End


for (int i=0; i < N; i++){
	for (int j=0; j < N; j++){
		B[i*N+j] = 0;
	}
}


clock_t after_init = clock();

hipProfilerStart();
// REQD: Step 2: Allocate device memory for A and B
// Start
uint8_t* d_A;
hipError_t err = hipMalloc((void**)&d_A, size);
printf("CUDA malloc d_A: %s\n",hipGetErrorString(err));
uint8_t* d_B;
err = hipMalloc((void**)&d_B, size);
printf("CUDA malloc d_B: %s\n",hipGetErrorString(err));

// End

// Step 3: Copy data from host memory to device memory
hipEvent_t start_memcpyh2d,stop_memcpyh2d;
hipEventCreate(&start_memcpyh2d);
hipEventCreate(&stop_memcpyh2d);


hipEventRecord(start_memcpyh2d);
// REQD: Code to Copy Data from Host to Device
// Start
err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
printf("CUDA Memcpy A->Ad: %s\n",hipGetErrorString(err));

printf("MEMCPY of %d bytes from Host to Device \n",size);

// End
hipEventRecord(stop_memcpyh2d);

float mseconds1 = 0.0;
hipEventElapsedTime(&mseconds1,start_memcpyh2d,stop_memcpyh2d);
printf("Time of the MEMCPY of %d bytes: %2.3f ms\n",size,mseconds1);


// REQD: Setup the execution configuration (Use dim3 objects to setup the Blocks and Grid)
// Start

dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y);
dim3 dimGrid(GRID_DIM_X, GRID_DIM_Y);


// End

// Step 4: Launch the device computation
hipEvent_t start_kernel,stop_kernel;
hipEventCreate(&start_kernel);
hipEventCreate(&stop_kernel);

hipEventRecord(start_kernel);
// REQD: Write the kernel call with <<< >>> to launch the GPU device
// Start
SobelFilter<<<dimGrid, dimBlock>>>(d_A, d_B);

// End
hipEventRecord(stop_kernel);

// Step 4.5: Wait for the Kernel to complete the workload
hipError_t errk = hipDeviceSynchronize();
hipEventSynchronize(stop_kernel);

float mksec = 0.0;
hipEventElapsedTime(&mksec,start_kernel,stop_kernel);
printf("Time to complete CUDA Sobel kernel %d size: %2.3f ms\n",N,mksec);
printf("CUDA kernel launch: %s\n",hipGetErrorString(errk));


// Step 5: Read results (matrix B) from the device
hipEvent_t start_memcpyd2h,stop_memcpyd2h;
hipEventCreate(&start_memcpyd2h);
hipEventCreate(&stop_memcpyd2h);
hipEventRecord(start_memcpyd2h);
// REQD: Write the code to copy result back to the Host (CPU)
// Start
err = hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
printf("CUDA Memcpy d_B->B: %s\n",hipGetErrorString(err));

// End
hipEventRecord(stop_memcpyd2h);

float mseconds2 = 0.0;
hipEventElapsedTime(&mseconds2,start_memcpyd2h,stop_memcpyd2h);
printf("Time of the MEMCPY of %d bytes : %2.3f ms\n",size,mseconds2);

// REQD: Step 6: Free device memory
// Start
hipFree(d_A);
hipFree(d_B);

// End

hipProfilerStop();

printf("Execution time for initialization(msec) = %d\n",(((after_init-before_init)*1000)/CLOCKS_PER_SEC));
printf("Execution time for CUDA Sobel Filter(msec)= %2.3f\n",mksec);

write_image_file(argv[2], B);

// Free host memory
free(A);
free(B);

return 0;
}	






